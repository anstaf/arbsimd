#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace testing {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto* _pp_var_a __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_b __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_c __attribute__((unused)) = params_.state_vars[2];\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        _pp_var_a[tid_] =  0.20000000000000001;
        _pp_var_b[tid_] =  0.29999999999999999;
        _pp_var_c[tid_] =  0.5;
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type t_16_, t_15_, t_13_, t_12_, t_11_, t_9_, t_7_, t_6_, t_5_, t_4_, t_10_, p_2_, j_8_, j_4_, j_6_, f_2_, j_2_, t_3_, j_5_, t_17_, t_8_, j_1_, t_14_, j_3_, f1, f_0_, j_7_, f_1_, r0, t_2_, f0, j_0_, p_0_, t_1_, r1, p_1_, t_0_;
        p_0_ = _pp_var_a[tid_];
        t_0_ = _pp_var_a[tid_];
        p_1_ = _pp_var_b[tid_];
        t_1_ = _pp_var_b[tid_];
        p_2_ = _pp_var_c[tid_];
        t_2_ = _pp_var_c[tid_];
        f0 =  2.0;
        r0 =  1.0;
        f1 =  3.0;
        r1 =  0.;
        f_0_ = t_0_-(p_0_+ -1.0*(t_1_*(t_0_*f0)-t_2_*r0)*dt);
        f_1_ = t_1_-(p_1_+( -1.0*(t_1_*(t_0_*f0)-t_2_*r0)+(t_2_*f1-t_1_*r1))*dt);
        f_2_ = t_2_-(p_2_+(t_1_*(t_0_*f0)-t_2_*r0+ -1.0*(t_2_*f1-t_1_*r1))*dt);
        j_0_ =  1.0- -1.0*(t_1_*f0)*dt;
        j_1_ =  -( -1.0*(t_0_*f0)*dt);
        j_2_ =  -( -1.0* -r0*dt);
        j_3_ =  -( -1.0*(t_1_*f0)*dt);
        j_4_ =  1.0-( -1.0*(t_0_*f0)+ -r1)*dt;
        j_5_ =  -(( -1.0* -r0+f1)*dt);
        j_6_ =  -(t_1_*f0*dt);
        j_7_ =  -((t_0_*f0+ -1.0* -r1)*dt);
        j_8_ =  1.0-( -r0+ -1.0*f1)*dt;
        t_3_ = j_8_*j_0_-j_2_*j_6_;
        t_4_ = j_8_*j_1_-j_2_*j_7_;
        t_5_ = j_8_*f_0_-j_2_*f_2_;
        t_6_ = j_8_*j_3_-j_5_*j_6_;
        t_7_ = j_8_*j_4_-j_5_*j_7_;
        t_8_ = j_8_*f_1_-j_5_*f_2_;
        t_9_ = t_7_*t_3_-t_4_*t_6_;
        t_10_ = t_7_*t_5_-t_4_*t_8_;
        t_11_ = t_7_*j_6_-j_7_*t_6_;
        t_12_ = t_7_*j_8_;
        t_13_ = t_7_*f_2_-j_7_*t_8_;
        t_14_ = t_9_*t_7_;
        t_15_ = t_9_*t_8_-t_6_*t_10_;
        t_16_ = t_9_*t_12_;
        t_17_ = t_9_*t_13_-t_11_*t_10_;
        t_0_ = t_0_-t_10_/t_9_;
        t_1_ = t_1_-t_15_/t_14_;
        t_2_ = t_2_-t_17_/t_16_;
        f_0_ = t_0_-(p_0_+ -1.0*(t_1_*(t_0_*f0)-t_2_*r0)*dt);
        f_1_ = t_1_-(p_1_+( -1.0*(t_1_*(t_0_*f0)-t_2_*r0)+(t_2_*f1-t_1_*r1))*dt);
        f_2_ = t_2_-(p_2_+(t_1_*(t_0_*f0)-t_2_*r0+ -1.0*(t_2_*f1-t_1_*r1))*dt);
        j_0_ =  1.0- -1.0*(t_1_*f0)*dt;
        j_1_ =  -( -1.0*(t_0_*f0)*dt);
        j_2_ =  -( -1.0* -r0*dt);
        j_3_ =  -( -1.0*(t_1_*f0)*dt);
        j_4_ =  1.0-( -1.0*(t_0_*f0)+ -r1)*dt;
        j_5_ =  -(( -1.0* -r0+f1)*dt);
        j_6_ =  -(t_1_*f0*dt);
        j_7_ =  -((t_0_*f0+ -1.0* -r1)*dt);
        j_8_ =  1.0-( -r0+ -1.0*f1)*dt;
        t_3_ = j_8_*j_0_-j_2_*j_6_;
        t_4_ = j_8_*j_1_-j_2_*j_7_;
        t_5_ = j_8_*f_0_-j_2_*f_2_;
        t_6_ = j_8_*j_3_-j_5_*j_6_;
        t_7_ = j_8_*j_4_-j_5_*j_7_;
        t_8_ = j_8_*f_1_-j_5_*f_2_;
        t_9_ = t_7_*t_3_-t_4_*t_6_;
        t_10_ = t_7_*t_5_-t_4_*t_8_;
        t_11_ = t_7_*j_6_-j_7_*t_6_;
        t_12_ = t_7_*j_8_;
        t_13_ = t_7_*f_2_-j_7_*t_8_;
        t_14_ = t_9_*t_7_;
        t_15_ = t_9_*t_8_-t_6_*t_10_;
        t_16_ = t_9_*t_12_;
        t_17_ = t_9_*t_13_-t_11_*t_10_;
        t_0_ = t_0_-t_10_/t_9_;
        t_1_ = t_1_-t_15_/t_14_;
        t_2_ = t_2_-t_17_/t_16_;
        f_0_ = t_0_-(p_0_+ -1.0*(t_1_*(t_0_*f0)-t_2_*r0)*dt);
        f_1_ = t_1_-(p_1_+( -1.0*(t_1_*(t_0_*f0)-t_2_*r0)+(t_2_*f1-t_1_*r1))*dt);
        f_2_ = t_2_-(p_2_+(t_1_*(t_0_*f0)-t_2_*r0+ -1.0*(t_2_*f1-t_1_*r1))*dt);
        j_0_ =  1.0- -1.0*(t_1_*f0)*dt;
        j_1_ =  -( -1.0*(t_0_*f0)*dt);
        j_2_ =  -( -1.0* -r0*dt);
        j_3_ =  -( -1.0*(t_1_*f0)*dt);
        j_4_ =  1.0-( -1.0*(t_0_*f0)+ -r1)*dt;
        j_5_ =  -(( -1.0* -r0+f1)*dt);
        j_6_ =  -(t_1_*f0*dt);
        j_7_ =  -((t_0_*f0+ -1.0* -r1)*dt);
        j_8_ =  1.0-( -r0+ -1.0*f1)*dt;
        t_3_ = j_8_*j_0_-j_2_*j_6_;
        t_4_ = j_8_*j_1_-j_2_*j_7_;
        t_5_ = j_8_*f_0_-j_2_*f_2_;
        t_6_ = j_8_*j_3_-j_5_*j_6_;
        t_7_ = j_8_*j_4_-j_5_*j_7_;
        t_8_ = j_8_*f_1_-j_5_*f_2_;
        t_9_ = t_7_*t_3_-t_4_*t_6_;
        t_10_ = t_7_*t_5_-t_4_*t_8_;
        t_11_ = t_7_*j_6_-j_7_*t_6_;
        t_12_ = t_7_*j_8_;
        t_13_ = t_7_*f_2_-j_7_*t_8_;
        t_14_ = t_9_*t_7_;
        t_15_ = t_9_*t_8_-t_6_*t_10_;
        t_16_ = t_9_*t_12_;
        t_17_ = t_9_*t_13_-t_11_*t_10_;
        t_0_ = t_0_-t_10_/t_9_;
        t_1_ = t_1_-t_15_/t_14_;
        t_2_ = t_2_-t_17_/t_16_;
        _pp_var_a[tid_] = t_0_;
        _pp_var_b[tid_] = t_1_;
        _pp_var_c[tid_] = t_2_;
    }
}

} // namespace

void mechanism_test3_kin_diff_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 3}, block_dim>>>(*p);
}

void mechanism_test3_kin_diff_gpu_compute_currents_(arb_mechanism_ppack* p) {}

void mechanism_test3_kin_diff_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_test3_kin_diff_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_test3_kin_diff_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_test3_kin_diff_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace testing
