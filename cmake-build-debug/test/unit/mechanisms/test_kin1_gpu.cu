#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace testing {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto _pp_var_tau __attribute__((unused)) = params_.globals[0];\
auto* _pp_var_a __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_b __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_v __attribute__((unused)) = params_.state_vars[2];\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        _pp_var_a[tid_] =  0.01;
        _pp_var_b[tid_] =  0.;
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type t_3_, t_0_, a_3_, t_2_, a_2_, a_1_, t_1_, a_0_;
        a_0_ =  1.0- -1.0*( 0.66666666666666663/_pp_var_tau)*dt;
        a_1_ =  -( -1.0* -( 0.33333333333333331/_pp_var_tau)*dt);
        a_2_ =  -( 0.66666666666666663/_pp_var_tau*dt);
        a_3_ =  1.0- -( 0.33333333333333331/_pp_var_tau)*dt;
        t_0_ = a_3_*a_0_-a_1_*a_2_;
        t_1_ = a_3_*_pp_var_a[tid_]-a_1_*_pp_var_b[tid_];
        t_2_ = t_0_*a_3_;
        t_3_ = t_0_*_pp_var_b[tid_]-a_2_*t_1_;
        _pp_var_a[tid_] = t_1_/t_0_;
        _pp_var_b[tid_] = t_3_/t_2_;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type current_ = 0;
        arb_value_type il = 0;
        il = _pp_var_a[tid_];
        current_ = il;
        _pp_var_vec_i[node_indexi_] = fma(10.0*_pp_var_weight[tid_], current_, _pp_var_vec_i[node_indexi_]);
    }
}

} // namespace

void mechanism_test_kin1_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 2}, block_dim>>>(*p);
}

void mechanism_test_kin1_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_test_kin1_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_test_kin1_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_test_kin1_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_test_kin1_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace testing
