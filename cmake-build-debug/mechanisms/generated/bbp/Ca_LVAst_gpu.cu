#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace bbp_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto* _pp_var_m __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_h __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_gCa_LVAstbar __attribute__((unused)) = params_.parameters[0];\
auto& _pp_var_ion_ca __attribute__((unused)) = params_.ion_states[0];\
auto* _pp_var_ion_ca_index __attribute__((unused)) = params_.ion_states[0].index;\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        _pp_var_m[tid_] =  1.0/( 1.0+exp( -(v+ 40.0)* 0.16666666666666666));
        _pp_var_h[tid_] =  1.0/( 1.0+exp((v+ 90.0)* 0.15625));
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type a_1_, ba_0_, a_0_, qt, ba_1_, ll3_, ll1_, mTau, mInf, hTau, hInf, ll0_, ll2_;
        ll3_ =  0.;
        ll2_ =  0.;
        ll1_ =  0.;
        ll0_ =  0.;
        qt =  2.952882641412121;
        mInf =  1.0/( 1.0+exp( -(v+ 40.0)* 0.16666666666666666));
        hInf =  1.0/( 1.0+exp((v+ 90.0)* 0.15625));
        mTau =  5.0+ 20.0/( 1.0+exp((v+ 35.0)* 0.20000000000000001));
        hTau =  20.0+ 50.0/( 1.0+exp((v+ 50.0)* 0.14285714285714285));
        a_0_ =  -1.0*qt/mTau;
        ba_0_ = mInf*qt/mTau/a_0_;
        ll0_ = a_0_*dt;
        ll1_ = ( 1.0+ 0.5*ll0_)/( 1.0- 0.5*ll0_);
        _pp_var_m[tid_] =  -ba_0_+(_pp_var_m[tid_]+ba_0_)*ll1_;
        a_1_ =  -1.0*qt/hTau;
        ba_1_ = hInf*qt/hTau/a_1_;
        ll2_ = a_1_*dt;
        ll3_ = ( 1.0+ 0.5*ll2_)/( 1.0- 0.5*ll2_);
        _pp_var_h[tid_] =  -ba_1_+(_pp_var_h[tid_]+ba_1_)*ll3_;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_ca_indexi_ = _pp_var_ion_ca_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type conductivity_ = 0;
        arb_value_type current_ = 0;
        arb_value_type eca = _pp_var_ion_ca.reversal_potential[ion_ca_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type ica = 0;
        ica = _pp_var_gCa_LVAstbar[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_h[tid_]*(v-eca);
        current_ = ica;
        conductivity_ = _pp_var_gCa_LVAstbar[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_h[tid_];
        _pp_var_vec_g[node_indexi_] = fma(10.0*_pp_var_weight[tid_], conductivity_, _pp_var_vec_g[node_indexi_]);
        _pp_var_vec_i[node_indexi_] = fma(10.0*_pp_var_weight[tid_], current_, _pp_var_vec_i[node_indexi_]);
        _pp_var_ion_ca.current_density[ion_ca_indexi_] = fma(10.0*_pp_var_weight[tid_], ica, _pp_var_ion_ca.current_density[ion_ca_indexi_]);
    }
}

} // namespace

void mechanism_Ca_LVAst_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 2}, block_dim>>>(*p);
}

void mechanism_Ca_LVAst_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_Ca_LVAst_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_Ca_LVAst_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_Ca_LVAst_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_Ca_LVAst_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace bbp_catalogue
} // namespace arb
