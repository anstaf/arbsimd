#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace bbp_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto _pp_var_zTau __attribute__((unused)) = params_.globals[0];\
auto* _pp_var_z __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_gSK_E2bar __attribute__((unused)) = params_.parameters[0];\
auto& _pp_var_ion_k __attribute__((unused)) = params_.ion_states[0];\
auto* _pp_var_ion_k_index __attribute__((unused)) = params_.ion_states[0].index;\
auto& _pp_var_ion_ca __attribute__((unused)) = params_.ion_states[1];\
auto* _pp_var_ion_ca_index __attribute__((unused)) = params_.ion_states[1].index;\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_ca_indexi_ = _pp_var_ion_ca_index[tid_];
        arb_value_type cai = _pp_var_ion_ca.internal_concentration[ion_ca_indexi_];
        if (cai< 9.9999999999999995e-08) {
            _pp_var_z[tid_] =  0.;
        }
        else {
            _pp_var_z[tid_] =  1.0/( 1.0+pow( 0.00042999999999999999/cai,  4.7999999999999998));
        }
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_ca_indexi_ = _pp_var_ion_ca_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type cai = _pp_var_ion_ca.internal_concentration[ion_ca_indexi_];
        arb_value_type a_0_, ll0_, ll2_, ll1_, b_0_;
        ll2_ =  0.;
        ll1_ =  0.;
        ll0_ =  0.;
        if (cai< 9.9999999999999995e-08) {
            ll0_ =  0.;
        }
        else {
            ll0_ =  1.0/( 1.0+pow( 0.00042999999999999999/cai,  4.7999999999999998));
        }
        a_0_ = _pp_var_zTau;
        b_0_ = ll0_;
        ll1_ =  -dt/a_0_;
        ll2_ = ( 1.0+ 0.5*ll1_)/( 1.0- 0.5*ll1_);
        _pp_var_z[tid_] = b_0_+(_pp_var_z[tid_]-b_0_)*ll2_;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_k_indexi_ = _pp_var_ion_k_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type conductivity_ = 0;
        arb_value_type current_ = 0;
        arb_value_type ek = _pp_var_ion_k.reversal_potential[ion_k_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type ik = 0;
        ik = _pp_var_gSK_E2bar[tid_]*_pp_var_z[tid_]*(v-ek);
        current_ = ik;
        conductivity_ = _pp_var_gSK_E2bar[tid_]*_pp_var_z[tid_];
        _pp_var_vec_g[node_indexi_] = fma(10.0*_pp_var_weight[tid_], conductivity_, _pp_var_vec_g[node_indexi_]);
        _pp_var_vec_i[node_indexi_] = fma(10.0*_pp_var_weight[tid_], current_, _pp_var_vec_i[node_indexi_]);
        _pp_var_ion_k.current_density[ion_k_indexi_] = fma(10.0*_pp_var_weight[tid_], ik, _pp_var_ion_k.current_density[ion_k_indexi_]);
    }
}

} // namespace

void mechanism_SK_E2_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 1}, block_dim>>>(*p);
}

void mechanism_SK_E2_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_SK_E2_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_SK_E2_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_SK_E2_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_SK_E2_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace bbp_catalogue
} // namespace arb
