#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace bbp_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto* _pp_var_m __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_h __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_gNap_Et2bar __attribute__((unused)) = params_.parameters[0];\
auto& _pp_var_ion_na __attribute__((unused)) = params_.ion_states[0];\
auto* _pp_var_ion_na_index __attribute__((unused)) = params_.ion_states[0].index;\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        _pp_var_m[tid_] =  1.0/( 1.0+exp( -(v+ 52.600000000000001)* 0.21739130434782611));
        _pp_var_h[tid_] =  1.0/( 1.0+exp((v+ 48.799999999999997)* 0.10000000000000001));
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type ba_0_, qt, a_0_, mRho, mInf, mBeta, a_1_, mAlpha, hRho, hInf, ll2_, hBeta, ll0_, ba_1_, ll3_, ll1_, hAlpha;
        ll3_ =  0.;
        ll2_ =  0.;
        ll1_ =  0.;
        ll0_ =  0.;
        qt =  2.952882641412121;
        mInf =  1.0/( 1.0+exp( -(v+ 52.600000000000001)* 0.21739130434782611));
        mAlpha =  1.0920000000000001*exprelr( -(v+ 38.0)* 0.16666666666666666);
        mBeta =  0.74399999999999999*exprelr((v+ 38.0)* 0.16666666666666666);
        mRho = mAlpha+mBeta;
        hInf =  1.0/( 1.0+exp((v+ 48.799999999999997)* 0.10000000000000001));
        hAlpha =  1.3334399999999999e-05*exprelr((v+ 17.0)* 0.21598272138228941);
        hBeta =  1.8252199999999998e-05*exprelr( -(v+ 64.400000000000006)* 0.38022813688212931);
        hRho = hAlpha+hBeta;
        a_0_ = qt* -1.0*mRho* 0.16666666666666666;
        ba_0_ = qt*mInf*mRho* 0.16666666666666666/a_0_;
        ll0_ = a_0_*dt;
        ll1_ = ( 1.0+ 0.5*ll0_)/( 1.0- 0.5*ll0_);
        _pp_var_m[tid_] =  -ba_0_+(_pp_var_m[tid_]+ba_0_)*ll1_;
        a_1_ = qt* -1.0*hRho;
        ba_1_ = qt*hInf*hRho/a_1_;
        ll2_ = a_1_*dt;
        ll3_ = ( 1.0+ 0.5*ll2_)/( 1.0- 0.5*ll2_);
        _pp_var_h[tid_] =  -ba_1_+(_pp_var_h[tid_]+ba_1_)*ll3_;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_na_indexi_ = _pp_var_ion_na_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type conductivity_ = 0;
        arb_value_type current_ = 0;
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type ena = _pp_var_ion_na.reversal_potential[ion_na_indexi_];
        arb_value_type ina = 0;
        ina = _pp_var_gNap_Et2bar[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_h[tid_]*(v-ena);
        current_ = ina;
        conductivity_ = _pp_var_gNap_Et2bar[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_h[tid_];
        _pp_var_vec_g[node_indexi_] = fma(10.0*_pp_var_weight[tid_], conductivity_, _pp_var_vec_g[node_indexi_]);
        _pp_var_vec_i[node_indexi_] = fma(10.0*_pp_var_weight[tid_], current_, _pp_var_vec_i[node_indexi_]);
        _pp_var_ion_na.current_density[ion_na_indexi_] = fma(10.0*_pp_var_weight[tid_], ina, _pp_var_ion_na.current_density[ion_na_indexi_]);
    }
}

} // namespace

void mechanism_Nap_Et2_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 2}, block_dim>>>(*p);
}

void mechanism_Nap_Et2_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_Nap_Et2_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_Nap_Et2_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_Nap_Et2_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_Nap_Et2_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace bbp_catalogue
} // namespace arb
