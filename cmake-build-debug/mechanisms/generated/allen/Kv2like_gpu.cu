#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace allen_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto* _pp_var_m __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_h1 __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_h2 __attribute__((unused)) = params_.state_vars[2];\
auto* _pp_var_gbar __attribute__((unused)) = params_.parameters[0];\
auto& _pp_var_ion_k __attribute__((unused)) = params_.ion_states[0];\
auto* _pp_var_ion_k_index __attribute__((unused)) = params_.ion_states[0].index;\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type mBeta, mAlpha, hInf, ll0_, ll1_;
        ll1_ =  0.;
        ll0_ =  0.;
        ll0_ =  43.0-v;
        ll1_ =  11.0*exprelr(ll0_* 0.090909090909090912);
        mAlpha =  0.12*ll1_;
        mBeta =  0.02*exp( -(v+ 1.27)* 0.0083333333333333332);
        hInf =  1.0/( 1.0+exp((v+ 58.0)* 0.090909090909090912));
        _pp_var_m[tid_] = mAlpha/(mAlpha+mBeta);
        _pp_var_h1[tid_] = hInf;
        _pp_var_h2[tid_] = hInf;
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type celsius = _pp_var_temperature_degC[node_indexi_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type a_1_, a_2_, a_0_, ll4_, qt, mRat, mBeta, hInf, ba_0_, ba_1_, ll3_, ll6_, h1Rat, ll1_, ba_2_, ll2_, mAlpha, h2Rat, ll5_, ll0_, ll7_;
        ll7_ =  0.;
        ll6_ =  0.;
        ll5_ =  0.;
        ll4_ =  0.;
        ll3_ =  0.;
        ll2_ =  0.;
        ll1_ =  0.;
        ll0_ =  0.;
        qt = pow( 2.2999999999999998, (celsius- 21.0)* 0.10000000000000001);
        ll0_ =  43.0-v;
        ll1_ =  11.0*exprelr(ll0_* 0.090909090909090912);
        mAlpha =  0.12*ll1_;
        mBeta =  0.02*exp( -(v+ 1.27)* 0.0083333333333333332);
        mRat =  0.40000000000000002*qt*(mAlpha+mBeta);
        hInf =  1.0/( 1.0+exp((v+ 58.0)* 0.090909090909090912));
        h1Rat = qt/( 360.0+( 1010.0+ 23.699999999999999*(v+ 54.0))*exp(pow( -((v+ 75.0)* 0.020833333333333332),  2.0)));
        h2Rat = qt/( 2350.0+ 1380.0*exp( -0.010999999999999999*v)- 210.0*exp( -0.029999999999999999*v));
        if (h2Rat< 0.) {
            h2Rat =  0.001;
        }
        a_0_ =  -mRat;
        ba_0_ =  0.40000000000000002*qt*mAlpha/a_0_;
        ll2_ = a_0_*dt;
        ll3_ = ( 1.0+ 0.5*ll2_)/( 1.0- 0.5*ll2_);
        _pp_var_m[tid_] =  -ba_0_+(_pp_var_m[tid_]+ba_0_)*ll3_;
        a_1_ =  -1.0*h1Rat;
        ba_1_ = hInf*h1Rat/a_1_;
        ll4_ = a_1_*dt;
        ll5_ = ( 1.0+ 0.5*ll4_)/( 1.0- 0.5*ll4_);
        _pp_var_h1[tid_] =  -ba_1_+(_pp_var_h1[tid_]+ba_1_)*ll5_;
        a_2_ =  -1.0*h2Rat;
        ba_2_ = hInf*h2Rat/a_2_;
        ll6_ = a_2_*dt;
        ll7_ = ( 1.0+ 0.5*ll6_)/( 1.0- 0.5*ll6_);
        _pp_var_h2[tid_] =  -ba_2_+(_pp_var_h2[tid_]+ba_2_)*ll7_;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_k_indexi_ = _pp_var_ion_k_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type conductivity_ = 0;
        arb_value_type current_ = 0;
        arb_value_type ek = _pp_var_ion_k.reversal_potential[ion_k_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type ik = 0;
        ik =  0.5*_pp_var_gbar[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*(_pp_var_h1[tid_]+_pp_var_h2[tid_])*(v-ek);
        current_ = ik;
        conductivity_ =  0.5*_pp_var_gbar[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*(_pp_var_h1[tid_]+_pp_var_h2[tid_]);
        _pp_var_vec_g[node_indexi_] = fma(10.0*_pp_var_weight[tid_], conductivity_, _pp_var_vec_g[node_indexi_]);
        _pp_var_vec_i[node_indexi_] = fma(10.0*_pp_var_weight[tid_], current_, _pp_var_vec_i[node_indexi_]);
        _pp_var_ion_k.current_density[ion_k_indexi_] = fma(10.0*_pp_var_weight[tid_], ik, _pp_var_ion_k.current_density[ion_k_indexi_]);
    }
}

} // namespace

void mechanism_Kv2like_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 3}, block_dim>>>(*p);
}

void mechanism_Kv2like_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_Kv2like_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_Kv2like_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_Kv2like_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_Kv2like_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace allen_catalogue
} // namespace arb
