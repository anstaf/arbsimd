#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace allen_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto _pp_var_vshift __attribute__((unused)) = params_.globals[0];\
auto _pp_var_mTauF __attribute__((unused)) = params_.globals[1];\
auto _pp_var_hTauF __attribute__((unused)) = params_.globals[2];\
auto* _pp_var_m __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_h __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_gbar __attribute__((unused)) = params_.parameters[0];\
auto& _pp_var_ion_k __attribute__((unused)) = params_.ion_states[0];\
auto* _pp_var_ion_k_index __attribute__((unused)) = params_.ion_states[0].index;\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        _pp_var_m[tid_] =  1.0/( 1.0+exp( -(v+ 47.0-_pp_var_vshift)* 0.034482758620689655));
        _pp_var_h[tid_] =  1.0/( 1.0+exp((v+ 66.0-_pp_var_vshift)* 0.10000000000000001));
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type celsius = _pp_var_temperature_degC[node_indexi_];
        arb_value_type a_1_, ba_0_, a_0_, hRat, qt, mInf, mRat, hInf, ll0_, ba_1_, ll3_, ll1_, ll2_;
        ll3_ =  0.;
        ll2_ =  0.;
        ll1_ =  0.;
        ll0_ =  0.;
        qt = pow( 2.2999999999999998, (celsius- 21.0)* 0.10000000000000001);
        mInf =  1.0/( 1.0+exp( -(v+ 47.0-_pp_var_vshift)* 0.034482758620689655));
        hInf =  1.0/( 1.0+exp((v+ 66.0-_pp_var_vshift)* 0.10000000000000001));
        mRat = qt/( 0.34000000000000002+_pp_var_mTauF* 0.92000000000000004*exp(pow( -((v+ 71.0-_pp_var_vshift)* 0.016949152542372881),  2.0)));
        hRat = qt/( 8.0+_pp_var_hTauF* 49.0*exp(pow( -((v+ 73.0-_pp_var_vshift)* 0.043478260869565216),  2.0)));
        a_0_ =  -1.0*mRat;
        ba_0_ = mInf*mRat/a_0_;
        ll0_ = a_0_*dt;
        ll1_ = ( 1.0+ 0.5*ll0_)/( 1.0- 0.5*ll0_);
        _pp_var_m[tid_] =  -ba_0_+(_pp_var_m[tid_]+ba_0_)*ll1_;
        a_1_ =  -1.0*hRat;
        ba_1_ = hInf*hRat/a_1_;
        ll2_ = a_1_*dt;
        ll3_ = ( 1.0+ 0.5*ll2_)/( 1.0- 0.5*ll2_);
        _pp_var_h[tid_] =  -ba_1_+(_pp_var_h[tid_]+ba_1_)*ll3_;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_k_indexi_ = _pp_var_ion_k_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type conductivity_ = 0;
        arb_value_type current_ = 0;
        arb_value_type ek = _pp_var_ion_k.reversal_potential[ion_k_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type ik = 0;
        ik = _pp_var_gbar[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_h[tid_]*(v-ek);
        current_ = ik;
        conductivity_ = _pp_var_gbar[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_h[tid_];
        _pp_var_vec_g[node_indexi_] = fma(10.0*_pp_var_weight[tid_], conductivity_, _pp_var_vec_g[node_indexi_]);
        _pp_var_vec_i[node_indexi_] = fma(10.0*_pp_var_weight[tid_], current_, _pp_var_vec_i[node_indexi_]);
        _pp_var_ion_k.current_density[ion_k_indexi_] = fma(10.0*_pp_var_weight[tid_], ik, _pp_var_ion_k.current_density[ion_k_indexi_]);
    }
}

} // namespace

void mechanism_K_T_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 2}, block_dim>>>(*p);
}

void mechanism_K_T_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_K_T_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_K_T_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_K_T_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_K_T_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace allen_catalogue
} // namespace arb
