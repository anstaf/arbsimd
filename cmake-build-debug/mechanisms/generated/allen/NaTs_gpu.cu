#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace allen_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto _pp_var_malphaF __attribute__((unused)) = params_.globals[0];\
auto _pp_var_mbetaF __attribute__((unused)) = params_.globals[1];\
auto _pp_var_mvhalf __attribute__((unused)) = params_.globals[2];\
auto _pp_var_mk __attribute__((unused)) = params_.globals[3];\
auto _pp_var_halphaF __attribute__((unused)) = params_.globals[4];\
auto _pp_var_hbetaF __attribute__((unused)) = params_.globals[5];\
auto _pp_var_hvhalf __attribute__((unused)) = params_.globals[6];\
auto _pp_var_hk __attribute__((unused)) = params_.globals[7];\
auto* _pp_var_m __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_h __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_v __attribute__((unused)) = params_.state_vars[2];\
auto* _pp_var_g __attribute__((unused)) = params_.state_vars[3];\
auto* _pp_var_celsius __attribute__((unused)) = params_.state_vars[4];\
auto* _pp_var_mInf __attribute__((unused)) = params_.state_vars[5];\
auto* _pp_var_mTau __attribute__((unused)) = params_.state_vars[6];\
auto* _pp_var_mAlpha __attribute__((unused)) = params_.state_vars[7];\
auto* _pp_var_mBeta __attribute__((unused)) = params_.state_vars[8];\
auto* _pp_var_hInf __attribute__((unused)) = params_.state_vars[9];\
auto* _pp_var_hTau __attribute__((unused)) = params_.state_vars[10];\
auto* _pp_var_hAlpha __attribute__((unused)) = params_.state_vars[11];\
auto* _pp_var_hBeta __attribute__((unused)) = params_.state_vars[12];\
auto* _pp_var_gbar __attribute__((unused)) = params_.parameters[0];\
auto& _pp_var_ion_na __attribute__((unused)) = params_.ion_states[0];\
auto* _pp_var_ion_na_index __attribute__((unused)) = params_.ion_states[0].index;\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__device__
void rates(arb_mechanism_ppack params_, int tid_, arb_value_type v) {
    PPACK_IFACE_BLOCK;
    arb_value_type ll1_, ll2_, ll3_, ll6_, ll4_, qt, ll5_, ll0_, ll7_;
    ll7_ =  0.;
    ll6_ =  0.;
    ll5_ =  0.;
    ll4_ =  0.;
    ll3_ =  0.;
    ll2_ =  0.;
    ll1_ =  0.;
    ll0_ =  0.;
    qt = pow( 2.2999999999999998, (_pp_var_celsius[tid_]- 23.0)* 0.10000000000000001);
    ll0_ =  -(v-_pp_var_mvhalf);
    if (abs(ll0_/_pp_var_mk)< 9.9999999999999995e-07) {
        ll1_ = _pp_var_mk*( 1.0-ll0_/_pp_var_mk* 0.5);
    }
    else {
        ll1_ = ll0_/(exp(ll0_/_pp_var_mk)- 1.0);
    }
    _pp_var_mAlpha[tid_] = _pp_var_malphaF*ll1_;
    ll2_ = v-_pp_var_mvhalf;
    if (abs(ll2_/_pp_var_mk)< 9.9999999999999995e-07) {
        ll3_ = _pp_var_mk*( 1.0-ll2_/_pp_var_mk* 0.5);
    }
    else {
        ll3_ = ll2_/(exp(ll2_/_pp_var_mk)- 1.0);
    }
    _pp_var_mBeta[tid_] = _pp_var_mbetaF*ll3_;
    _pp_var_mInf[tid_] = _pp_var_mAlpha[tid_]/(_pp_var_mAlpha[tid_]+_pp_var_mBeta[tid_]);
    _pp_var_mTau[tid_] =  1.0/(_pp_var_mAlpha[tid_]+_pp_var_mBeta[tid_])/qt;
    ll4_ = v-_pp_var_hvhalf;
    if (abs(ll4_/_pp_var_hk)< 9.9999999999999995e-07) {
        ll5_ = _pp_var_hk*( 1.0-ll4_/_pp_var_hk* 0.5);
    }
    else {
        ll5_ = ll4_/(exp(ll4_/_pp_var_hk)- 1.0);
    }
    _pp_var_hAlpha[tid_] = _pp_var_halphaF*ll5_;
    ll6_ =  -(v-_pp_var_hvhalf);
    if (abs(ll6_/_pp_var_hk)< 9.9999999999999995e-07) {
        ll7_ = _pp_var_hk*( 1.0-ll6_/_pp_var_hk* 0.5);
    }
    else {
        ll7_ = ll6_/(exp(ll6_/_pp_var_hk)- 1.0);
    }
    _pp_var_hBeta[tid_] = _pp_var_hbetaF*ll7_;
    _pp_var_hInf[tid_] = _pp_var_hAlpha[tid_]/(_pp_var_hAlpha[tid_]+_pp_var_hBeta[tid_]);
    _pp_var_hTau[tid_] =  1.0/(_pp_var_hAlpha[tid_]+_pp_var_hBeta[tid_])/qt;
}

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        rates(params_, tid_, v);
        _pp_var_m[tid_] = _pp_var_mInf[tid_];
        _pp_var_h[tid_] = _pp_var_hInf[tid_];
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type a_1_, a_0_, b_0_, ll0_, ll1_, ll2_, b_1_, ll3_;
        ll3_ =  0.;
        ll2_ =  0.;
        ll1_ =  0.;
        ll0_ =  0.;
        rates(params_, tid_, v);
        a_0_ = _pp_var_mTau[tid_];
        b_0_ = _pp_var_mInf[tid_];
        ll0_ =  -dt/a_0_;
        ll1_ = ( 1.0+ 0.5*ll0_)/( 1.0- 0.5*ll0_);
        _pp_var_m[tid_] = b_0_+(_pp_var_m[tid_]-b_0_)*ll1_;
        a_1_ = _pp_var_hTau[tid_];
        b_1_ = _pp_var_hInf[tid_];
        ll2_ =  -dt/a_1_;
        ll3_ = ( 1.0+ 0.5*ll2_)/( 1.0- 0.5*ll2_);
        _pp_var_h[tid_] = b_1_+(_pp_var_h[tid_]-b_1_)*ll3_;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_na_indexi_ = _pp_var_ion_na_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type conductivity_ = 0;
        arb_value_type current_ = 0;
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type ena = _pp_var_ion_na.reversal_potential[ion_na_indexi_];
        arb_value_type ina = 0;
        _pp_var_g[tid_] = _pp_var_gbar[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_m[tid_]*_pp_var_h[tid_];
        ina = _pp_var_g[tid_]*(v-ena);
        current_ = ina;
        conductivity_ = _pp_var_g[tid_];
        _pp_var_vec_g[node_indexi_] = fma(10.0*_pp_var_weight[tid_], conductivity_, _pp_var_vec_g[node_indexi_]);
        _pp_var_vec_i[node_indexi_] = fma(10.0*_pp_var_weight[tid_], current_, _pp_var_vec_i[node_indexi_]);
        _pp_var_ion_na.current_density[ion_na_indexi_] = fma(10.0*_pp_var_weight[tid_], ina, _pp_var_ion_na.current_density[ion_na_indexi_]);
    }
}

} // namespace

void mechanism_NaTs_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 2}, block_dim>>>(*p);
}

void mechanism_NaTs_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_NaTs_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_NaTs_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_NaTs_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_NaTs_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace allen_catalogue
} // namespace arb
